/* Copyright (c) 1993-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hipblas.h>
#include <sys/time.h>
#include <pthread.h>
// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
   }
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
   if (stat != HIPRAND_STATUS_SUCCESS) {
      fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
   }
}



__global__ void print_gpu_values(half *a, half *b, half *c_half, float *c_float)
{
    printf("a %f, b %f, c_half %f, c_float %f\n", __half2float(*a), __half2float(*b), __half2float(*c_half), *c_float);
}

__global__ void convertFp32ToFp16 (half *out, float *in, int n, float scale) {
   int idx = blockDim.x * blockIdx.x + threadIdx.x;
   if (idx < n) {
      if(scale > 32768.0)
          out[idx] = (half)(in[idx]/scale);
      else
      out[idx] = in[idx];
   }
}


int main(int argc, char* argv[]) {
   float *a_fp32, *A;
   float *b_fp32, *B;
   float scale;
   half *a_fp16;
   half *b_fp16;

   float *c;
   float *c_cublas_gemmEx;

   float *c_host_cublas;
   float *c_host_cublasCublasGemmEx;


   float alpha = 1.0f;
   float beta = 0.0f;
   int mode = 0,i,ret,current;
   FILE *fin;

   int MATRIX_M=10240;
   int MATRIX_N=10240;
   int MATRIX_K=10240;
   
   hipblasHandle_t cublasHandle;
   

   
   hipEvent_t startcublasEX;
   hipEvent_t stopcublasEX;



   hipEvent_t startcublasCublasGemmEx;
   hipEvent_t stopcublasCublasGemmEx;
   
   struct timeval time_start, time_end, total_start, total_end;
   int init_time;
   gettimeofday(&time_start, NULL);
   hipFree(0);

   cudaErrCheck(hipEventCreate(&startcublasEX));
   cudaErrCheck(hipEventCreate(&stopcublasEX));

   cudaErrCheck(hipEventCreate(&startcublasCublasGemmEx));
   cudaErrCheck(hipEventCreate(&stopcublasCublasGemmEx));
   A = (float*)malloc(MATRIX_M * MATRIX_K * sizeof(float));
   B = (float*)malloc(MATRIX_K * MATRIX_N * sizeof(float));
    if(mode == 0)
    {
        fin = fopen(argv[1], "r");
        for (i = 0; i < (MATRIX_M*MATRIX_K); i++) {
            ret = fscanf(fin,"%f",&A[i]); 
        }
        fclose(fin);
        fin = fopen(argv[2], "r");
        for (i = 0; i < (MATRIX_K * MATRIX_N); i++) {
            ret = fscanf(fin,"%f",&B[i]);
        }
        fclose(fin);
    }
    else
    {
        fin = fopen(argv[1], "rb");
        for (i = 0; i < (MATRIX_M*MATRIX_K); i++) {
            ret = fread(&current, 1, sizeof(int), fin);
            A[i] = (float)current;
        }
        fclose(fin);
        fin = fopen(argv[2], "rb");
        for (i = 0; i < (MATRIX_K * MATRIX_N); i++) {
            ret = fread(&current, 1, sizeof(int), fin);
            B[i] = (float)current;
        }
        fclose(fin);
    }

   gettimeofday(&total_start, NULL);

   cublasErrCheck(hipblasCreate(&cublasHandle));

   // Use tensor cores
   cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));

  	gettimeofday(&total_end, NULL);
        init_time = ((total_end.tv_sec * 1000000 + total_end.tv_usec) - (total_start.tv_sec * 1000000 + total_start.tv_usec));
	fprintf(stderr,"cublasCreate(2)  %d\n",init_time);
   MATRIX_M = atoi(argv[3]);
   MATRIX_K = MATRIX_M;
   MATRIX_N = MATRIX_M;
   cudaErrCheck(hipMalloc((void**)&a_fp32, MATRIX_M * MATRIX_K * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&b_fp32, MATRIX_K * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&a_fp16, MATRIX_M * MATRIX_K * sizeof(half)));
   cudaErrCheck(hipMalloc((void**)&b_fp16, MATRIX_K * MATRIX_N * sizeof(half)));

   cudaErrCheck(hipMalloc((void**)&c, MATRIX_M * MATRIX_N * sizeof(float)));

   cudaErrCheck(hipMalloc((void**)&c_cublas_gemmEx, MATRIX_M * MATRIX_N * sizeof(float)));

   c_host_cublas = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));
   c_host_cublasCublasGemmEx = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));

   cudaErrCheck(hipMemcpy(a_fp32, A, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));
   cudaErrCheck(hipMemcpy(b_fp32, B, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyHostToDevice)); 
   scale=3.4028234664e+38;
   
   // curand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
   convertFp32ToFp16 <<< (MATRIX_M * MATRIX_K + 255) / 256, 256 >>> (a_fp16, a_fp32, MATRIX_M * MATRIX_K, scale);
   convertFp32ToFp16 <<< (MATRIX_K * MATRIX_N + 255) / 256, 256 >>> (b_fp16, b_fp32, MATRIX_K * MATRIX_N, scale);

   cudaErrCheck(hipMemcpy(c_cublas_gemmEx, c, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToDevice));

  	gettimeofday(&time_end, NULL);
        init_time = ((time_end.tv_sec * 1000000 + time_end.tv_usec) - (time_start.tv_sec * 1000000 + time_start.tv_usec));
	fprintf(stderr,"Before GEMM %d\n",init_time);

   printf("\nM = %d, N = %d, K = %d. alpha = %f, beta = %f\n\n", MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);

   // Now using cuBLAS but not tensor
   printf("Running with cuBLAS (GemmEX)...\n");
   cudaErrCheck(hipEventRecord(startcublasCublasGemmEx));

   cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                MATRIX_M, MATRIX_N, MATRIX_K, 
                &alpha,
                a_fp16, HIP_R_16F, MATRIX_M,
                b_fp16, HIP_R_16F, MATRIX_K,
                &beta, 
                c_cublas_gemmEx, HIP_R_32F, MATRIX_M,
                HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP));
   cudaErrCheck(hipEventRecord(stopcublasCublasGemmEx));

   // Error checking
   cudaErrCheck(hipMemcpy(c_host_cublasCublasGemmEx, c_cublas_gemmEx, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));


      float cublasTime;

      cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublasCublasGemmEx, stopcublasCublasGemmEx));
      printf("cublas hipblasGemmEx took %f ms, GFLOS: %lf\n", cublasTime, (double)((double)MATRIX_M * (double)MATRIX_N*(double)MATRIX_K)*2/((double)cublasTime*1000000));


   cudaErrCheck(hipEventDestroy(startcublasCublasGemmEx));
   cudaErrCheck(hipEventDestroy(stopcublasCublasGemmEx));


   cudaErrCheck(hipFree(a_fp32));
   cudaErrCheck(hipFree(b_fp32));
   cudaErrCheck(hipFree(a_fp16));
   cudaErrCheck(hipFree(b_fp16));

   cudaErrCheck(hipFree(c));
   cudaErrCheck(hipFree(c_cublas_gemmEx));
   
   free(c_host_cublas);

   cudaErrCheck(hipDeviceReset());
   return 0;
}


