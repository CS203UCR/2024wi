/* Copyright (c) 1993-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hipblas.h>
#include <sys/time.h>
// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
   }
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
   if (stat != HIPRAND_STATUS_SUCCESS) {
      fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
   }
}


int main(int argc, char* argv[]) {
   float *a_fp32, *A;
   float *b_fp32, *B;

   float *c;
   float *c_cublas;
   float *c_host_cublas;

   float alpha = 1.0f;
   float beta = 0.0f;
   int mode = 0,i,ret,current;
   FILE *fin;

   int MATRIX_M=10240;
   int MATRIX_N=10240;
   int MATRIX_K=10240;
   
   hipblasHandle_t cublasHandle;


   hipEvent_t startcublas;
   hipEvent_t stopcublas;

   struct timeval time_start, time_end, total_start, total_end;
   int init_time;
   gettimeofday(&time_start, NULL);
   hipFree(0);

   
   cudaErrCheck(hipEventCreate(&startcublas));
   cudaErrCheck(hipEventCreate(&stopcublas));


   A = (float*)malloc(MATRIX_M * MATRIX_K * sizeof(float));
   B = (float*)malloc(MATRIX_K * MATRIX_N * sizeof(float));
    if(mode == 0)
    {
        fin = fopen(argv[1], "r");
        for (i = 0; i < (MATRIX_M*MATRIX_K); i++) {
            ret = fscanf(fin,"%f",&A[i]); 
        }
        fclose(fin);
        fin = fopen(argv[2], "r");
        for (i = 0; i < (MATRIX_K * MATRIX_N); i++) {
            ret = fscanf(fin,"%f",&B[i]);
        }
        fclose(fin);
    }
    else
    {
        fin = fopen(argv[1], "rb");
        for (i = 0; i < (MATRIX_M*MATRIX_K); i++) {
            ret = fread(&current, 1, sizeof(int), fin);
            A[i] = (float)current;
        }
        fclose(fin);
        fin = fopen(argv[2], "rb");
        for (i = 0; i < (MATRIX_K * MATRIX_N); i++) {
            ret = fread(&current, 1, sizeof(int), fin);
            B[i] = (float)current;
        }
        fclose(fin);
    }

   gettimeofday(&total_start, NULL);

   cublasErrCheck(hipblasCreate(&cublasHandle));
   // Use tensor cores
   cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_DEFAULT_MATH));
   gettimeofday(&total_end, NULL);
   init_time = ((total_end.tv_sec * 1000000 + total_end.tv_usec) - (total_start.tv_sec * 1000000 + total_start.tv_usec));
	fprintf(stderr,"cublasCreate(2)  %d\n",init_time);
   MATRIX_M = atoi(argv[3]);
   MATRIX_K = MATRIX_M;
   MATRIX_N = MATRIX_M;
   cudaErrCheck(hipMalloc((void**)&a_fp32, MATRIX_M * MATRIX_K * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&b_fp32, MATRIX_K * MATRIX_N * sizeof(float)));

   cudaErrCheck(hipMalloc((void**)&c, MATRIX_M * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&c_cublas, MATRIX_M * MATRIX_N * sizeof(float)));

   c_host_cublas = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));

   cudaErrCheck(hipMemcpy(a_fp32, A, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));
   cudaErrCheck(hipMemcpy(b_fp32, B, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyHostToDevice)); 


   cudaErrCheck(hipMemcpy(c_cublas, c, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToDevice));

   gettimeofday(&time_end, NULL);
   init_time = ((time_end.tv_sec * 1000000 + time_end.tv_usec) - (time_start.tv_sec * 1000000 + time_start.tv_usec));
   fprintf(stderr,"Before GEMM %d\n",init_time);


   printf("\nM = %d, N = %d, K = %d. alpha = %f, beta = %f\n\n", MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);

   printf("Running with sgemm...\n");
   cudaErrCheck(hipEventRecord(startcublas));
   hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, MATRIX_M, MATRIX_N, MATRIX_K, &alpha, a_fp32, MATRIX_M, b_fp32, MATRIX_N, &beta, c_cublas, MATRIX_K);
   cudaErrCheck(hipEventRecord(stopcublas));
   

   cudaErrCheck(hipMemcpy(c_host_cublas, c_cublas, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));

      float cublasTime;
      cudaErrCheck(hipEventSynchronize(stopcublas));
      cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas));
      printf("cublas took %f ms, GFLOS: %lf\n", cublasTime,(double)((double)MATRIX_M * (double)MATRIX_N*(double)MATRIX_K)*2/((double)cublasTime*1000000));


   cudaErrCheck(hipEventDestroy(startcublas));             
   cudaErrCheck(hipEventDestroy(stopcublas));
   
   cudaErrCheck(hipFree(a_fp32));
   cudaErrCheck(hipFree(b_fp32));

   cudaErrCheck(hipFree(c));
   cudaErrCheck(hipFree(c_cublas));
   
   free(c_host_cublas);

   cudaErrCheck(hipDeviceReset());
   return 0;
}


